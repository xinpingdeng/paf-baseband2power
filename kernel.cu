#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include "kernel.cuh"
#include "baseband2power.cuh"
#include "cudautil.cuh"


/*
  This kernel is used to :
  1. unpack the incoming data reading from ring buffer;
  2. reorder the order from TFTFP to PFT;
  3. detect the data
*/
__global__ void unpack_detect_kernel(int64_t *dbuf_in,  float *dbuf_rt1)
{
  size_t loc_in, loc_rt1;
  int64_t tmp;
  int16_t p1x, p1y, p2x, p2y;
  
  /* 
     Loc for the input array, it is in continuous order, it is in (STREAM_BUF_NDFSTP)T(NCHK_NIC)F(NSAMP_DF)T(NCHAN_CHK)F(NPOL_SAMP)P order
     This is for entire setting, since gridDim.z =1 and blockDim.z = 1, we can simply it to the latter format;
     Becareful here, if these number are not 1, we need to use a different format;
   */
  //loc_in = blockIdx.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z +
  //  blockIdx.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z +
  //  blockIdx.z * blockDim.x * blockDim.y * blockDim.z +
  //  threadIdx.x * blockDim.y * blockDim.z +
  //  threadIdx.y * blockDim.z +
  //  threadIdx.z;
  loc_in = blockIdx.x * gridDim.y * blockDim.x * blockDim.y +
    blockIdx.y * blockDim.x * blockDim.y +
    threadIdx.x * blockDim.y +
    threadIdx.y;
  tmp = BSWAP_64(dbuf_in[loc_in]);
  
  // Put the data into FT order  
  loc_rt1 = blockIdx.y * gridDim.x * blockDim.x * blockDim.y +
    threadIdx.y * gridDim.x * blockDim.x +
    blockIdx.x * blockDim.x +
    threadIdx.x;

  p1x = (int16_t)((tmp & 0x000000000000ffffULL));  
  p1y = (int16_t)((tmp & 0x00000000ffff0000ULL) >> 16);
  p2x = (int16_t)((tmp & 0x0000ffff00000000ULL) >> 32);
  p2y = (int16_t)((tmp & 0xffff000000000000ULL) >> 48);
  
  dbuf_rt1[loc_rt1] = p1x * p1x + p1y * p1y + p2x * p2x + p2y * p2y;
}


/*
  This kernel will get the sum of all elements in dbuf_rt1, which is the buffer for each stream
 */
__global__ void sum_kernel(float *dbuf_rt1, float *dbuf_rt2)
{
  extern __shared__ float sum_sdata[];
  size_t tid, loc, s;
  
  tid = threadIdx.x;
  loc = blockIdx.x * gridDim.y * (blockDim.x * 2) +
    blockIdx.y * (blockDim.x * 2) +
    threadIdx.x;
  sum_sdata[tid] = dbuf_rt1[loc];
  __syncthreads();

  /* do reduction in shared mem */
  for (s=blockDim.x/2; s>0; s>>=1)
    {
      if (tid < s)
	sum_sdata[tid] += sum_sdata[tid + s];
      __syncthreads();
    }

  /* write result of this block to global mem */
  if (tid == 0)
    dbuf_rt2[blockIdx.x * gridDim.y + blockIdx.y] = sum_sdata[0];
}
