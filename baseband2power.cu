#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <time.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>
#include <math.h>

#include "multilog.h"
#include "baseband2power.cuh"
#include "paf_baseband2power.cuh"
#include "cudautil.cuh"
#include "kernel.cuh"

extern multilog_t *runtime_log;

int init_baseband2power(conf_t *conf)
{
  CudaSafeCall(hipSetDevice(conf->device_id));
  size_t curbufsz;
  uint64_t block_id;
  
  ipcbuf_t *db = NULL;
  if(conf->nchan_out != (NCHAN_CHK * NCHK_NIC))
    {
      multilog(runtime_log, LOG_ERR, "Channel number mismatch\n");
      fprintf(stderr, "The number of channel is not match %d != %d, which happens at \"%s\", line [%d].\n",conf->nchan_out, NCHAN_CHK * NCHK_NIC, __FILE__, __LINE__);
    }
  conf->bufin_size  = conf->rbufin_ndf * NCHAN_CHK * NCHK_NIC * NSAMP_DF * NPOL_SAMP * NDIM_POL * NBYTE_IN;
  conf->bufrt_size  = conf->rbufin_ndf * NCHAN_CHK * NCHK_NIC * NSAMP_DF * NPOL_SAMP * NDIM_POL * NBYTE_RT;
  conf->bufout_size = NCHAN_CHK * NCHK_NIC * NBYTE_OUT;
  
  conf->nsamp_in  = conf->rbufin_ndf * NCHAN_CHK * NCHK_NIC * NSAMP_DF;
  conf->nsamp_rt  = conf->nsamp_in;
  conf->nsamp_out = conf->nchan_out;

  CudaSafeCall(hipMalloc((void **)&conf->dbuf_in, conf->bufin_size));
  CudaSafeCall(hipMalloc((void **)&conf->dbuf_out, conf->bufout_size));
  CudaSafeCall(hipMalloc((void **)&conf->buf_rt, conf->bufrt_size));
  
  /* Prepare the setup of kernels */
  conf->gridsize_unpack.x = conf->rbufin_ndf;
  conf->gridsize_unpack.y = NCHK_NIC;
  conf->gridsize_unpack.z = 1;
  conf->blocksize_unpack.x = NSAMP_DF; 
  conf->blocksize_unpack.y = NCHAN_CHK;
  conf->blocksize_unpack.z = 1;

  /* Attach to input ring buffer */
  conf->hdu_in = dada_hdu_create(runtime_log);
  dada_hdu_set_key(conf->hdu_in, conf->key_in);
  if(dada_hdu_connect(conf->hdu_in) < 0)
    {
      multilog(runtime_log, LOG_ERR, "could not connect to hdu\n");
      fprintf(stderr, "Can not connect to hdu, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }  
  db = (ipcbuf_t *) conf->hdu_in->data_block;
  if(ipcbuf_get_bufsz(db) != conf->bufin_size)
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }

  /* registers the existing host memory range for use by CUDA */
  dada_cuda_dbregister(conf->hdu_in);
  
  if(ipcbuf_get_bufsz(conf->hdu_in->header_block) != DADA_HDR_SIZE)    // This number should match
    {
      multilog(runtime_log, LOG_ERR, "Header buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  
  /* make ourselves the read client */
  if(dada_hdu_lock_read(conf->hdu_in) < 0)
    {
      multilog(runtime_log, LOG_ERR, "open_hdu: could not lock write\n");
      fprintf(stderr, "Error locking HDU, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  
  /* Prepare output ring buffer */
  conf->hdu_out = dada_hdu_create(runtime_log);
  dada_hdu_set_key(conf->hdu_out, conf->key_out);
  if(dada_hdu_connect(conf->hdu_out) < 0)
    {
      multilog(runtime_log, LOG_ERR, "could not connect to hdu\n");
      fprintf(stderr, "Can not connect to hdu, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  db = (ipcbuf_t *) conf->hdu_out->data_block;
  if(ipcbuf_get_bufsz(db) != conf->bufout_size)  
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  
  if(ipcbuf_get_bufsz(conf->hdu_out->header_block) != DADA_HDR_SIZE)    // This number should match
    {
      multilog(runtime_log, LOG_ERR, "Header buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }  
  /* make ourselves the write client */
  if(dada_hdu_lock_write(conf->hdu_out) < 0)
    {
      multilog(runtime_log, LOG_ERR, "open_hdu: could not lock write\n");
      fprintf(stderr, "Error locking HDU, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  
  if(conf->sod)
    {      
      if(ipcbuf_enable_sod(db, 0, 0) < 0)  // We start at the beginning
  	{
	  multilog(runtime_log, LOG_ERR, "Can not write data before start, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
  	  fprintf(stderr, "Can not write data before start, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
  	  return EXIT_FAILURE;
  	}
    }
  else
    {
      if(ipcbuf_disable_sod(db) < 0)
  	{
	  multilog(runtime_log, LOG_ERR, "Can not write data before start, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
  	  fprintf(stderr, "Can not write data before start, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
  	  return EXIT_FAILURE;
  	}
    }
  
  /* Register header */
  if(register_header(conf))
    {
      multilog(runtime_log, LOG_ERR, "header register failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "header register failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }

  conf->hdu_out->data_block->curbuf = ipcio_open_block_write(conf->hdu_out->data_block, &block_id);   /* Open buffer to write */
  conf->hdu_in->data_block->curbuf = ipcio_open_block_read(conf->hdu_in->data_block, &curbufsz, &block_id);
    
  return EXIT_SUCCESS;
}

int destroy_process(conf_t conf)
{
  CudaSafeCall(hipSetDevice(conf.device_id));

  hipFree(conf.dbuf_in);
  hipFree(conf.dbuf_out);
  hipFree(conf.buf_rt);

  dada_cuda_dbunregister(conf.hdu_in);
  dada_hdu_unlock_read(conf.hdu_in);
  dada_hdu_unlock_write(conf.hdu_out);
  
  return EXIT_SUCCESS;
}

int do_baseband2power(conf_t conf)
{
  CudaSafeCall(hipSetDevice(conf.device_id));
  size_t curbufsz;
  uint64_t block_id;
  
#ifdef DEBUG
  struct timespec start, stop;
  double elapsed_time;
#endif
  
  while(conf.hdu_in->data_block->curbufsz == conf.bufin_size)
    {
#ifdef DEBUG
      clock_gettime(CLOCK_REALTIME, &start);
#endif

      CudaSafeCall(hipMemcpy(conf.dbuf_in, conf.hdu_in->data_block->curbuf, conf.bufin_size, hipMemcpyHostToDevice));
      
#ifdef DEBUG
      clock_gettime(CLOCK_REALTIME, &stop);
      elapsed_time = (stop.tv_sec - start.tv_sec) + (stop.tv_nsec - start.tv_nsec)/1000000000.0L;
      fprintf(stdout, "Elapsed time to copy %"PRIu64" bytes data is %f second.\n", conf.bufin_size, elapsed_time);
#endif
      
      if(ipcio_close_block_read(conf.hdu_in->data_block, conf.hdu_in->data_block->curbufsz)<0)
      	{
	  multilog (runtime_log, LOG_ERR, "close_buffer: ipcio_close_block_write failed\n");
	  fprintf(stderr, "close_buffer: ipcio_close_block_write failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	  return EXIT_FAILURE;
	}

      fprintf(stdout, "HERE\n");
      
      conf.hdu_in->data_block->curbuf = ipcio_open_block_read(conf.hdu_in->data_block, &curbufsz, &block_id);
      
      CudaSafeCall(hipMemcpy(conf.hdu_out->data_block->curbuf, conf.dbuf_out, conf.bufout_size, hipMemcpyDeviceToHost));

      if(ipcio_close_block_write(conf.hdu_out->data_block, conf.bufout_size)<0)
	{
	  multilog (runtime_log, LOG_ERR, "close_buffer: ipcio_close_block_write failed\n");
	  fprintf(stderr, "close_buffer: ipcio_close_block_write failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	  return EXIT_FAILURE;
	}

      conf.hdu_out->data_block->curbuf = ipcio_open_block_write(conf.hdu_out->data_block, &block_id);   /* Open buffer to write */     
    }
  
  return EXIT_SUCCESS;
}

int register_header(conf_t *conf)
{
  size_t hdrsz;
  
  conf->hdrbuf_in  = ipcbuf_get_next_read(conf->hdu_in->header_block, &hdrsz);
  if(hdrsz != DADA_HDR_SIZE)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Header size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if (!conf->hdrbuf_in)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Error getting header_buf, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  
  conf->hdrbuf_out = ipcbuf_get_next_write(conf->hdu_out->header_block);
  if (!conf->hdrbuf_out)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Error getting header_buf, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  memcpy(conf->hdrbuf_out, conf->hdrbuf_in, DADA_HDR_SIZE);
  if (ipcbuf_mark_filled (conf->hdu_out->header_block, DADA_HDR_SIZE) < 0)
    {
      multilog(runtime_log, LOG_ERR, "Could not mark filled header block\n");
      fprintf(stderr, "Error header_fill, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  
  if(ipcbuf_mark_cleared (conf->hdu_in->header_block))  // We are the only one reader, so that we can clear it after read;
    {
      multilog(runtime_log, LOG_ERR, "Could not clear header block\n");
      fprintf(stderr, "Error header_clear, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  
  return EXIT_SUCCESS;
}