#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <time.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>
#include <math.h>

#include "multilog.h"
#include "baseband2power.cuh"
#include "paf_baseband2power.cuh"
#include "cudautil.cuh"
#include "kernel.cuh"

extern multilog_t *runtime_log;

int init_baseband2power(conf_t *conf)
{
  CudaSafeCall(hipSetDevice(conf->device_id));
  size_t curbufsz;
  uint64_t block_id;
  
  ipcbuf_t *db = NULL;
  if(conf->nchan_out != (NCHAN_CHK * NCHK_NIC))
    {
      multilog(runtime_log, LOG_ERR, "Channel number mismatch\n");
      fprintf(stderr, "The number of channel is not match %d != %d, which happens at \"%s\", line [%d].\n",conf->nchan_out, NCHAN_CHK * NCHK_NIC, __FILE__, __LINE__);
    }
  conf->bufin_size  = conf->bufin_ndf * NCHAN_CHK * NCHK_NIC * NSAMP_DF * NPOL_SAMP * NDIM_POL * NBYTE_IN;
  conf->bufrt_size  = conf->bufin_ndf * NCHAN_CHK * NCHK_NIC * NSAMP_DF * NBYTE_RT;
  conf->bufout_size = NCHAN_CHK * NCHK_NIC * NBYTE_OUT;
  
  conf->nsamp_in  = conf->bufin_ndf * NCHAN_CHK * NCHK_NIC * NSAMP_DF;
  conf->nsamp_rt  = conf->nsamp_in;
  conf->nsamp_out = conf->nchan_out;
  
  CudaSafeCall(hipMalloc((void **)&conf->dbuf_in, conf->bufin_size));
  CudaSafeCall(hipMalloc((void **)&conf->dbuf_out, conf->bufout_size));
  CudaSafeCall(hipMalloc((void **)&conf->buf_rt1, conf->bufrt_size));
  CudaSafeCall(hipMalloc((void **)&conf->buf_rt2, conf->bufrt_size));
  
  /* Prepare the setup of kernels */
  conf->gridsize_unpack_detect.x = conf->bufin_ndf;
  conf->gridsize_unpack_detect.y = NCHK_NIC;
  conf->gridsize_unpack_detect.z = 1;
  conf->blocksize_unpack_detect.x = NSAMP_DF; 
  conf->blocksize_unpack_detect.y = NCHAN_CHK;
  conf->blocksize_unpack_detect.z = 1;

  conf->gridsize_sum1.x = NCHK_NIC * NCHAN_CHK;
  conf->gridsize_sum1.y = conf->bufin_ndf * NSAMP_DF / (2 * BLKSZ_SUM1);
  conf->gridsize_sum1.z = 1;
  conf->blocksize_sum1.x = BLKSZ_SUM1;
  conf->blocksize_sum1.y = 1;
  conf->blocksize_sum1.z = 1;

  conf->gridsize_sum2.x = NCHK_NIC * NCHAN_CHK;
  conf->gridsize_sum2.y = 1;
  conf->gridsize_sum2.z = 1;
  conf->blocksize_sum2.x = conf->bufin_ndf * NSAMP_DF / (4 * BLKSZ_SUM1);
  conf->blocksize_sum2.y = 1;
  conf->blocksize_sum2.z = 1;

  /* Attach to input ring buffer */
  conf->hdu_in = dada_hdu_create(runtime_log);
  dada_hdu_set_key(conf->hdu_in, conf->key_in);
  if(dada_hdu_connect(conf->hdu_in) < 0)
    {
      multilog(runtime_log, LOG_ERR, "could not connect to hdu\n");
      fprintf(stderr, "Can not connect to hdu, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }  
  db = (ipcbuf_t *) conf->hdu_in->data_block;
  if(ipcbuf_get_bufsz(db) != conf->bufin_size)
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }

  /* registers the existing host memory range for use by CUDA */
  dada_cuda_dbregister(conf->hdu_in);
  
  if(ipcbuf_get_bufsz(conf->hdu_in->header_block) != DADA_HDR_SIZE)    // This number should match
    {
      multilog(runtime_log, LOG_ERR, "Header buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  
  /* make ourselves the read client */
  if(dada_hdu_lock_read(conf->hdu_in) < 0)
    {
      multilog(runtime_log, LOG_ERR, "open_hdu: could not lock write\n");
      fprintf(stderr, "Error locking HDU, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  
  /* Prepare output ring buffer */
  conf->hdu_out = dada_hdu_create(runtime_log);
  dada_hdu_set_key(conf->hdu_out, conf->key_out);
  if(dada_hdu_connect(conf->hdu_out) < 0)
    {
      multilog(runtime_log, LOG_ERR, "could not connect to hdu\n");
      fprintf(stderr, "Can not connect to hdu, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  db = (ipcbuf_t *) conf->hdu_out->data_block;
  if(ipcbuf_get_bufsz(db) != conf->bufout_size)  
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  
  if(ipcbuf_get_bufsz(conf->hdu_out->header_block) != DADA_HDR_SIZE)    // This number should match
    {
      multilog(runtime_log, LOG_ERR, "Header buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }  
  /* make ourselves the write client */
  if(dada_hdu_lock_write(conf->hdu_out) < 0)
    {
      multilog(runtime_log, LOG_ERR, "open_hdu: could not lock write\n");
      fprintf(stderr, "Error locking HDU, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  
  if(conf->sod)
    {      
      if(ipcbuf_enable_sod(db, 0, 0) < 0)  // We start at the beginning
  	{
	  multilog(runtime_log, LOG_ERR, "Can not write data before start, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
  	  fprintf(stderr, "Can not write data before start, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
  	  return EXIT_FAILURE;
  	}
    }
  else
    {
      if(ipcbuf_disable_sod(db) < 0)
  	{
	  multilog(runtime_log, LOG_ERR, "Can not write data before start, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
  	  fprintf(stderr, "Can not write data before start, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
  	  return EXIT_FAILURE;
  	}
    }
  
  /* Register header */
  if(register_header(conf))
    {
      multilog(runtime_log, LOG_ERR, "header register failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "header register failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }

  conf->hdu_out->data_block->curbuf = ipcio_open_block_write(conf->hdu_out->data_block, &block_id);   /* Open buffer to write */
  conf->hdu_in->data_block->curbuf = ipcio_open_block_read(conf->hdu_in->data_block, &curbufsz, &block_id);
    
  return EXIT_SUCCESS;
}

int destroy_process(conf_t conf)
{
  CudaSafeCall(hipSetDevice(conf.device_id));

  hipFree(conf.dbuf_in);
  hipFree(conf.dbuf_out);
  hipFree(conf.buf_rt1);
  hipFree(conf.buf_rt2);

  dada_cuda_dbunregister(conf.hdu_in);
  dada_hdu_unlock_read(conf.hdu_in);
  dada_hdu_unlock_write(conf.hdu_out);
  
  return EXIT_SUCCESS;
}

int do_baseband2power(conf_t conf)
{
  CudaSafeCall(hipSetDevice(conf.device_id));
  size_t curbufsz;
  uint64_t block_id;
  dim3 gridsize_unpack_detect, blocksize_unpack_detect;
  dim3 gridsize_sum1, blocksize_sum1;
  dim3 gridsize_sum2, blocksize_sum2;

  gridsize_unpack_detect = conf.gridsize_unpack_detect;
  blocksize_unpack_detect = conf.blocksize_unpack_detect;
  gridsize_sum1 = conf.gridsize_sum1;
  blocksize_sum1 = conf.blocksize_sum1;
  gridsize_sum2 = conf.gridsize_sum2;
  blocksize_sum2 = conf.blocksize_sum2;
  
#ifdef DEBUG
  struct timespec start, stop;
  double elapsed_time;
#endif
  
  while(conf.hdu_in->data_block->curbufsz == conf.bufin_size)
    {
#ifdef DEBUG
      clock_gettime(CLOCK_REALTIME, &start);
#endif

      /* Copy data from host to device */
      CudaSafeCall(hipMemcpy(conf.dbuf_in, conf.hdu_in->data_block->curbuf, conf.bufin_size, hipMemcpyHostToDevice));
      
#ifdef DEBUG
      clock_gettime(CLOCK_REALTIME, &stop);
      elapsed_time = (stop.tv_sec - start.tv_sec) + (stop.tv_nsec - start.tv_nsec)/1000000000.0L;
      fprintf(stdout, "Elapsed time to copy %"PRIu64" bytes data is %f second.\n", conf.bufin_size, elapsed_time);
#endif

      /* Do processing here */
      unpack_detect_kernel<<<gridsize_unpack_detect, blocksize_unpack_detect, 0>>>(conf.dbuf_in, conf.buf_rt1);
      sum_kernel<<<gridsize_sum1, blocksize_sum1, blocksize_sum1.x * sizeof(float)>>>(conf.buf_rt1, conf.buf_rt2);
      sum_kernel<<<gridsize_sum2, blocksize_sum2, blocksize_sum2.x * sizeof(float)>>>(conf.buf_rt2, conf.dbuf_out);
      
      /* Close previous data block and open a new one to read */
      if(ipcio_close_block_read(conf.hdu_in->data_block, conf.hdu_in->data_block->curbufsz)<0)
      	{
	  multilog (runtime_log, LOG_ERR, "close_buffer: ipcio_close_block_write failed\n");
	  fprintf(stderr, "close_buffer: ipcio_close_block_write failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	  return EXIT_FAILURE;
	}
      conf.hdu_in->data_block->curbuf = ipcio_open_block_read(conf.hdu_in->data_block, &curbufsz, &block_id);

      /* Copy data from device to host */
      CudaSafeCall(hipMemcpy(conf.hdu_out->data_block->curbuf, conf.dbuf_out, conf.bufout_size, hipMemcpyDeviceToHost));
      /* Close previous data block and open a new one to write */
      if(ipcio_close_block_write(conf.hdu_out->data_block, conf.bufout_size)<0)
	{
	  multilog (runtime_log, LOG_ERR, "close_buffer: ipcio_close_block_write failed\n");
	  fprintf(stderr, "close_buffer: ipcio_close_block_write failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	  return EXIT_FAILURE;
	}

      conf.hdu_out->data_block->curbuf = ipcio_open_block_write(conf.hdu_out->data_block, &block_id);   /* Open buffer to write */     
    }
  
  return EXIT_SUCCESS;
}

int register_header(conf_t *conf)
{
  size_t hdrsz;
  double scale;
  conf->tsamp_out = NSAMP_DF * conf->bufin_ndf * TSAMP;
  //fprintf(stdout, "%f\n", conf->tsamp_out);
  
  conf->hdrbuf_in  = ipcbuf_get_next_read(conf->hdu_in->header_block, &hdrsz);
  if(hdrsz != DADA_HDR_SIZE)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Header size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if (!conf->hdrbuf_in)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Error getting header_buf, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if (ascii_header_get(conf->hdrbuf_in, "TSAMP", "%lf", &(conf->tsamp_in)) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "Error getting TSAMP, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "Error getting TSAMP, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }    
  if (ascii_header_get(conf->hdrbuf_in, "BYTES_PER_SECOND", "%lf", &(conf->bps_in)) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "Error getting BYTES_PER_SECOND, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "Error getting BYTES_PER_SECOND, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }    
  if (ascii_header_get(conf->hdrbuf_in, "FILE_SIZE", "%lf", &(conf->fsz_in)) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "Error getting FILE_SIZE, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "Error getting FILE_SIZE, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }

  scale = conf->tsamp_in / conf->tsamp_out;
  //fprintf(stdout, "%f\t%f\t%f\n", conf->tsamp_in, conf->tsamp_out, scale);
  conf->hdrbuf_out = ipcbuf_get_next_write(conf->hdu_out->header_block);
  if (!conf->hdrbuf_out)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Error getting header_buf, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  memcpy(conf->hdrbuf_out, conf->hdrbuf_in, DADA_HDR_SIZE);  
  if (ascii_header_set(conf->hdrbuf_out, "TSAMP", "%lf", conf->tsamp_out) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "Error getting TSAMP, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "Error getting TSAMP, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }    
  if (ascii_header_set(conf->hdrbuf_out, "BYTES_PER_SECOND", "%lf", conf->bps_in *scale) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "Error getting BYTES_PER_SECOND, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "Error getting BYTES_PER_SECOND, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }    
  if (ascii_header_set(conf->hdrbuf_out, "FILE_SIZE", "%lf", conf->fsz_in * scale) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "Error getting FILE_SIZE, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "Error getting FILE_SIZE, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }

  if (ipcbuf_mark_filled (conf->hdu_out->header_block, DADA_HDR_SIZE) < 0)
    {
      multilog(runtime_log, LOG_ERR, "Could not mark filled header block\n");
      fprintf(stderr, "Error header_fill, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  
  if(ipcbuf_mark_cleared (conf->hdu_in->header_block))  // We are the only one reader, so that we can clear it after read;
    {
      multilog(runtime_log, LOG_ERR, "Could not clear header block\n");
      fprintf(stderr, "Error header_clear, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  
  return EXIT_SUCCESS;
}