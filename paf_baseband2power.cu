#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <inttypes.h>

#include "multilog.h"
#include "baseband2power.cuh"
#include "paf_baseband2power.cuh"
#include "cudautil.cuh"
#include "kernel.cuh"

void usage ()
{
  fprintf (stdout,
	   "paf_baseband2power - To detect baseband data with original channels and average the detected data in time\n"
	   "\n"
	   "Usage: paf_process [options]\n"
	   " -a  Hexacdecimal shared memory key for incoming ring buffer\n"
	   " -b  Hexacdecimal shared memory key for outcoming ring buffer\n"
	   " -c  The name of the directory in which we will record the data\n"
	   " -d  The index of GPU\n"
	   " -h  show help\n");
}

multilog_t *runtime_log;

int main(int argc, char *argv[])
{
  int arg;
  FILE *fp_log = NULL;
  char log_fname[MSTR_LEN];
  conf_t conf;
  
  /* configuration from command line */
  while((arg=getopt(argc,argv,"a:b:c:d:h:")) != -1)
    {
      switch(arg)
	{
	case 'h':
	  usage();
	  return EXIT_FAILURE;
	  
	case 'a':	  	  
	  if (sscanf (optarg, "%x", &conf.key_in) != 1)
	    {
	      fprintf (stderr, "Could not parse key from %s, which happens at \"%s\", line [%d].\n", optarg, __FILE__, __LINE__);
	      return EXIT_FAILURE;
	    }
	  break;
	  	
	case 'b':	  	  
	  if (sscanf (optarg, "%x", &conf.key_out) != 1)
	    {
	      fprintf (stderr, "Could not parse key from %s, which happens at \"%s\", line [%d].\n", optarg, __FILE__, __LINE__);
	      return EXIT_FAILURE;
	    }
	  break;
	  		  
	case 'c':
	  sscanf(optarg, "%s", conf.dir);
	  break;

	case 'd':
	  sscanf(optarg, "%d", &conf.device_id);
	  break;	  
	}
    }
  
  /* Setup log interface */
  sprintf(log_fname, "%s/paf_baseband2power.log", conf.dir);
  fp_log = fopen(log_fname, "ab+");
  if(fp_log == NULL)
    {
      fprintf(stderr, "Can not open log file %s\n", log_fname);
      return EXIT_FAILURE;
    }
  runtime_log = multilog_open("paf_baseband2power", 1);
  multilog_add(runtime_log, fp_log);
  multilog(runtime_log, LOG_INFO, "START PAF_PROCESS\n");
  
  /* Here to make sure that if we only expose one GPU into docker container, we can get the right index of it */ 
  int deviceCount;
  CudaSafeCall(hipGetDeviceCount(&deviceCount));
  if(deviceCount == 1)
    conf.device_id = 0;

  return EXIT_SUCCESS;
}
