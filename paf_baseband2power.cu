#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <inttypes.h>

#include "multilog.h"
#include "baseband2power.cuh"
#include "paf_baseband2power.cuh"
#include "cudautil.cuh"
#include "kernel.cuh"

void usage ()
{
  fprintf (stdout,
	   "paf_baseband2power - To detect baseband data with original channels and average the detected data in time\n"
	   "\n"
	   "Usage: paf_process [options]\n"
	   " -a  Hexacdecimal shared memory key for incoming ring buffer\n"
	   " -b  Hexacdecimal shared memory key for outcoming ring buffer\n"
	   " -c  The name of the directory in which we will record the data\n"
	   " -d  The index of GPU\n"
	   " -e  The number of data fram steps of input ring buffer\n"
	   " -f  The number of channel of output data\n"
	   " -g  Enable the start-of-data\n"
	   " -h  show help\n");
}

multilog_t *runtime_log;

int main(int argc, char *argv[])
{
  int arg;
  FILE *fp_log = NULL;
  char log_fname[MSTR_LEN];
  conf_t conf;
  
  /* configuration from command line */
  while((arg=getopt(argc,argv,"a:b:c:d:e:f:g:h:")) != -1)
    {
      switch(arg)
	{
	case 'h':
	  usage();
	  return EXIT_FAILURE;
	  
	case 'a':	  	  
	  if (sscanf (optarg, "%x", &conf.key_in) != 1)
	    {
	      fprintf (stderr, "Could not parse key from %s, which happens at \"%s\", line [%d].\n", optarg, __FILE__, __LINE__);
	      return EXIT_FAILURE;
	    }
	  break;
	  	
	case 'b':	  	  
	  if (sscanf (optarg, "%x", &conf.key_out) != 1)
	    {
	      fprintf (stderr, "Could not parse key from %s, which happens at \"%s\", line [%d].\n", optarg, __FILE__, __LINE__);
	      return EXIT_FAILURE;
	    }
	  break;
	  		  
	case 'c':
	  sscanf(optarg, "%s", conf.dir);
	  break;

	case 'd':
	  sscanf(optarg, "%d", &conf.device_id);
	  break;

	case 'e':
	  sscanf(optarg, "%lf", &conf.bufin_ndf);
	  break;

	case 'f':
	  sscanf(optarg, "%d", &conf.nchan_out);
	  break;
	  
	case 'g':
	  sscanf(optarg, "%d", &conf.sod);
	  break;	  
	}
    }
  
  /* Setup log interface */
  sprintf(log_fname, "%s/paf_baseband2power.log", conf.dir);
  fp_log = fopen(log_fname, "ab+");
  if(fp_log == NULL)
    {
      fprintf(stderr, "Can not open log file %s\n", log_fname);
      return EXIT_FAILURE;
    }
  runtime_log = multilog_open("paf_baseband2power", 1);
  multilog_add(runtime_log, fp_log);
  multilog(runtime_log, LOG_INFO, "START PAF_BASEBAND2POWER\n");
  
  /* Here to make sure that if we only expose one GPU into docker container, we can get the right index of it */ 
  int deviceCount;
  CudaSafeCall(hipGetDeviceCount(&deviceCount));
  if(deviceCount == 1)
    conf.device_id = 0;

  /* Init process */
  init_baseband2power(&conf);
  
  /* Do process */
  do_baseband2power(conf);

  multilog(runtime_log, LOG_INFO, "FINISH PAF_BASEBAND2POWER\n\n");
  
  return EXIT_SUCCESS;
}
